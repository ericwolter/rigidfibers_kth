#include "hip/hip_runtime.h"
/*
 *  fibers - simulates slender fibers in a fluid.
 *  Copyright (C) 2014  Eric Wolter <eric.wolter@gmx.de>
 *
 *  This program is free software; you can redistribute it and/or
 *  modify it under the terms of the GNU General Public License
 *  as published by the Free Software Foundation; either version 2
 *  of the License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program; if not, write to the Free Software
 *  Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

#include <iostream>

#include "fiberopt.h"
#include "parameters.h"
#include "simulation.h"

#include "kernels/saxpy.cu"

int main(int argc, char *argv[])
{
    FiberArgs args = fiberopt(argc, argv,/* help */  1, /* version */ "v0.2.0");

    Configuration configuration = Parameters::parseConfigurationFiles(args.parameters, args.layout);

    Parameters::dump(configuration.parameters);

    // cleanup
    delete[] configuration.initial_positions;
    delete[] configuration.initial_orientations;

    Simulation simulation(configuration);

    bool running = true;
    unsigned long current_timestep = 0;
    do
    {
        std::cout << "     [CPU]      : Timestep " << current_timestep + 1 << " of " << configuration.parameters.num_timesteps << std::endl;
        simulation.step(current_timestep);

        current_timestep++;

        if(current_timestep >= configuration.parameters.num_timesteps) {
            running = false;
        }
    }
    while (running);

    int N = 1 << 20;
    float *x, *y, *d_x, *d_y;
    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy <<< (N + 255) / 256, 256 >>> (N, 2.0, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = max(maxError, abs(y[i] - 4.0f));
    std::cout << "Max error: " << maxError << std::endl;
}