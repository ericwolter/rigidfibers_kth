#include "hip/hip_runtime.h"
#ifndef FIBERS_UPDATE_VELOCITIES_KERNEL_
#define FIBERS_UPDATE_VELOCITIES_KERNEL_

#include "constants.cu"

__device__
    void compute_GV(const fiberuint j,
                const fiberfloat4 position_i,
                const fiberfloat4 orientation_i,
                const fiberfloat4 position_j,
                const fiberfloat4 orientation_j,
                const fiberfloat *coefficients,
                const fiberfloat4 external_force,
                const fiberfloat *quadrature_points,
                const fiberfloat *quadrature_weights,
                const fiberfloat *legendre_polynomials,
                fiberfloat *GF
                ) // @TODO better names
{
    for (fiberuint quadrature_index_i = 0; quadrature_index_i < TOTAL_NUMBER_OF_QUADRATURE_POINTS; ++quadrature_index_i)
    {
        GF[quadrature_index_i + 0 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] = 0.0f;
        GF[quadrature_index_i + 1 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] = 0.0f;
        GF[quadrature_index_i + 2 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] = 0.0f;

        fiberfloat4 position_on_fiber_i;
        position_on_fiber_i.x = position_i.x + quadrature_points[quadrature_index_i] * orientation_i.x;
        position_on_fiber_i.y = position_i.y + quadrature_points[quadrature_index_i] * orientation_i.y;
        position_on_fiber_i.z = position_i.z + quadrature_points[quadrature_index_i] * orientation_i.z;

        for (fiberuint quadrature_index_j = 0; quadrature_index_j < TOTAL_NUMBER_OF_QUADRATURE_POINTS; ++quadrature_index_j)
        {
            const fiberfloat quadrature_point = quadrature_points[quadrature_index_j];
            fiberfloat4 position_on_fiber_j;
            position_on_fiber_j.x = position_j.x + quadrature_point * orientation_j.x;
            position_on_fiber_j.y = position_j.y + quadrature_point * orientation_j.y;
            position_on_fiber_j.z = position_j.z + quadrature_point * orientation_j.z;

            fiberfloat4 difference;
            difference.x = position_on_fiber_i.x - position_on_fiber_j.x;
            difference.y = position_on_fiber_i.y - position_on_fiber_j.y;
            difference.z = position_on_fiber_i.z - position_on_fiber_j.z;

            const fiberfloat distance = sqrtf(difference.x * difference.x + difference.y * difference.y + difference.z * difference.z);

            // equation 10
            // Note:    The outer product of a vector with itself is always a symmetric matrix
            //          so to save computation we only compute the upper triangle.
            // TODO calculation can be optimized (i.e. not dividing by distance, simpifing etc.)
            const fiberfloat K11 = 1.0f / distance
                                   + (1.0f / distance) * (difference.x / distance) * (difference.x / distance)
                                   + 2.0f * SLENDERNESS * SLENDERNESS * ((1.0f / (distance * distance * distance))
                                           - (3.0f / (distance * distance * distance)) * ((difference.x / distance) * (difference.x / distance)));
            const fiberfloat K22 = 1.0f / distance
                                   + (1.0f / distance) * (difference.y / distance) * (difference.y / distance)
                                   + 2.0f * SLENDERNESS * SLENDERNESS * ((1.0f / (distance * distance * distance))
                                           - (3.0f / (distance * distance * distance)) * ((difference.y / distance) * (difference.y / distance)));
            const fiberfloat K33 = 1.0f / distance
                                   + (1.0f / distance) * (difference.z / distance) * (difference.z / distance)
                                   + 2.0f * SLENDERNESS * SLENDERNESS * ((1.0f / (distance * distance * distance))
                                           - (3.0f / (distance * distance * distance)) * ((difference.z / distance) * (difference.z / distance)));
            const fiberfloat K12 = (1.0f / distance) * (difference.x / distance) * (difference.y / distance)
                                   + 2.0f * SLENDERNESS * SLENDERNESS
                                   * (-3.0f / (distance * distance * distance)) * (difference.x / distance) * (difference.y / distance);

            const fiberfloat K13 = (1.0f / distance) * (difference.x / distance) * (difference.z / distance)
                                   + 2.0f * SLENDERNESS * SLENDERNESS
                                   * (-3.0f / (distance * distance * distance)) * (difference.x / distance) * (difference.z / distance);

            const fiberfloat K23 = (1.0f / distance) * (difference.y / distance) * (difference.z / distance)
                                   + 2.0f * SLENDERNESS * SLENDERNESS
                                   * (-3.0f / (distance * distance * distance)) * (difference.y / distance) * (difference.z / distance);

            const fiberfloat quadrature_weight = quadrature_weights[quadrature_index_j];

            fiberfloat4 force_on_fiber_j;
            force_on_fiber_j.x = 0.5f * external_force.x;
            force_on_fiber_j.y = 0.5f * external_force.y;
            force_on_fiber_j.z = 0.5f * external_force.z;

            for (fiberuint force_index = 0; force_index < NUMBER_OF_TERMS_IN_FORCE_EXPANSION; ++force_index)
            {
                const fiberfloat legendre_polynomial = legendre_polynomials[quadrature_index_j + force_index * TOTAL_NUMBER_OF_QUADRATURE_POINTS];
                fiberuint x_row_index = j * NUMBER_OF_TERMS_IN_FORCE_EXPANSION * DIMENSIONS + DIMENSIONS * force_index + 0;
                fiberuint y_row_index = j * NUMBER_OF_TERMS_IN_FORCE_EXPANSION * DIMENSIONS + DIMENSIONS * force_index + 1;
                fiberuint z_row_index = j * NUMBER_OF_TERMS_IN_FORCE_EXPANSION * DIMENSIONS + DIMENSIONS * force_index + 2;

                force_on_fiber_j.x += coefficients[x_row_index] * legendre_polynomial;
                force_on_fiber_j.y += coefficients[y_row_index] * legendre_polynomial;
                force_on_fiber_j.z += coefficients[z_row_index] * legendre_polynomial;
            }

            GF[quadrature_index_i + 0 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] += quadrature_weight * (K11 * force_on_fiber_j.x + K12 * force_on_fiber_j.y + K13 * force_on_fiber_j.z);
            GF[quadrature_index_i + 1 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] += quadrature_weight * (K12 * force_on_fiber_j.x + K22 * force_on_fiber_j.y + K23 * force_on_fiber_j.z);
            GF[quadrature_index_i + 2 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] += quadrature_weight * (K13 * force_on_fiber_j.x + K23 * force_on_fiber_j.y + K33 * force_on_fiber_j.z);
        }
    }
}

__global__ void update_velocities(
    const fiberfloat4 *positions,
    const fiberfloat4 *orientations,
    const fiberfloat *coefficients,
    fiberfloat4 *translational_velocities,
    fiberfloat4 *rotational_velocities,
    const fiberfloat *quadrature_points,
    const fiberfloat *quadrature_weights,
    const fiberfloat *legendre_polynomials
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= NUMBER_OF_FIBERS) return;

    const fiberfloat c  = logf(SLENDERNESS * SLENDERNESS * M_E);
    const fiberfloat d  = -c;

    const fiberfloat4 position_i = positions[i];
    const fiberfloat4 orientation_i = orientations[i];

    // @TODO Constant external force
    fiberfloat4 external_force;
    external_force.x = 0.0f;
    external_force.y = 0.0f;
    external_force.z = -1.0f;

    fiberfloat4 oriented_force;
    oriented_force.x = orientation_i.x * orientation_i.x * external_force.x + orientation_i.x * orientation_i.y * external_force.y + orientation_i.x * orientation_i.z * external_force.z;
    oriented_force.y = orientation_i.x * orientation_i.y * external_force.x + orientation_i.y * orientation_i.y * external_force.y + orientation_i.y * orientation_i.z * external_force.z;
    oriented_force.z = orientation_i.x * orientation_i.z * external_force.x + orientation_i.y * orientation_i.z * external_force.y + orientation_i.z * orientation_i.z * external_force.z;

    translational_velocities[i].x = 0.5f * ((d + 2.0f) * external_force.x + (d - 2.0f) * oriented_force.x);
    translational_velocities[i].y = 0.5f * ((d + 2.0f) * external_force.y + (d - 2.0f) * oriented_force.y);
    translational_velocities[i].z = 0.5f * ((d + 2.0f) * external_force.z + (d - 2.0f) * oriented_force.z);

    rotational_velocities[i].x = 0.0f;
    rotational_velocities[i].y = 0.0f;
    rotational_velocities[i].z = 0.0f;

    for (fiberuint j = 0; j < NUMBER_OF_FIBERS; ++j)
    {
        if (i == j) continue;

        const fiberfloat4 position_j = positions[j];
        const fiberfloat4 orientation_j = orientations[j];

        fiberfloat GF[24 * 3];
        compute_GV(j, position_i, orientation_i, position_j, orientation_j, coefficients, external_force, quadrature_points, quadrature_weights, legendre_polynomials, GF);

        fiberfloat TF1A0 = 0.0f;
        fiberfloat TF2A0 = 0.0f;
        fiberfloat TF3A0 = 0.0f;

        fiberfloat TF1A1 = 0.0f;
        fiberfloat TF2A1 = 0.0f;
        fiberfloat TF3A1 = 0.0f;

        for (fiberuint quadrature_index_i = 0; quadrature_index_i < TOTAL_NUMBER_OF_QUADRATURE_POINTS; ++quadrature_index_i)
        {
            const fiberfloat quadrature_weight = quadrature_weights[quadrature_index_i];
            const fiberfloat legendre_polynomial = legendre_polynomials[quadrature_index_i + 0 * TOTAL_NUMBER_OF_QUADRATURE_POINTS];

            TF1A0 += quadrature_weight * GF[quadrature_index_i + 0 * TOTAL_NUMBER_OF_QUADRATURE_POINTS];
            TF2A0 += quadrature_weight * GF[quadrature_index_i + 1 * TOTAL_NUMBER_OF_QUADRATURE_POINTS];
            TF3A0 += quadrature_weight * GF[quadrature_index_i + 2 * TOTAL_NUMBER_OF_QUADRATURE_POINTS];

            TF1A1 += quadrature_weight * GF[quadrature_index_i + 0 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] * legendre_polynomial;
            TF2A1 += quadrature_weight * GF[quadrature_index_i + 1 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] * legendre_polynomial;
            TF3A1 += quadrature_weight * GF[quadrature_index_i + 2 * TOTAL_NUMBER_OF_QUADRATURE_POINTS] * legendre_polynomial;
        }

        translational_velocities[i].x += 0.5f * TF1A0;
        translational_velocities[i].y += 0.5f * TF2A0;
        translational_velocities[i].z += 0.5f * TF3A0;

        rotational_velocities[i].x += 1.5f * (TF1A1 - (orientation_i.x * orientation_i.x * TF1A1 + orientation_i.x * orientation_i.y * TF2A1 + orientation_i.x * orientation_i.z * TF3A1));
        rotational_velocities[i].y += 1.5f * (TF2A1 - (orientation_i.x * orientation_i.y * TF1A1 + orientation_i.y * orientation_i.y * TF2A1 + orientation_i.y * orientation_i.z * TF3A1));
        rotational_velocities[i].z += 1.5f * (TF3A1 - (orientation_i.x * orientation_i.z * TF1A1 + orientation_i.y * orientation_i.z * TF2A1 + orientation_i.z * orientation_i.z * TF3A1));
    }

    translational_velocities[i].x /= d;
    translational_velocities[i].y /= d;
    translational_velocities[i].z /= d;

    rotational_velocities[i].x /= d;
    rotational_velocities[i].y /= d;
    rotational_velocities[i].z /= d;
}

#endif //FIBERS_UPDATE_VELOCITIES_KERNEL_
